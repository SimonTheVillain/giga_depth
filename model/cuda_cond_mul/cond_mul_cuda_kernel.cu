#include "hip/hip_runtime.h"

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>
#include <chrono>
//#define FORCE_DOUBLE

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
//gpuErrchk( hipPeekAtLastError() );
//gpuErrchk( hipDeviceSynchronize() );
namespace {



//extremely slow kernel... TODO: delete!!!
/*
template <typename scalar_t>
__global__ void cond_mul_cuda_forward_kernel(
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds, //indices are in int32 datatype
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> bias,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output) {
    //printf("i hate you\n");

    const int ind = blockIdx.x * blockDim.x + threadIdx.x;
    const int m = weights.size(1);
    const int n = weights.size(2);
    if(ind >= input.size(0)){
        return;
    }
    int ind_w = inds[ind];
    for(size_t i = 0; i<n ; i++){
        scalar_t accu = bias[ind_w][0][i];

        for(size_t j = 0; j<m; j++){
            //printf("weights %f input %f \n",weights[ind_w][j][i], input[ind][j]);
            accu += weights[ind_w][j][i] * input[ind][j];
        }

        output[ind][i] = accu;

    }
}
*/


//pretty trivial kernel that has as many threads in the x-dimension of a block as output channels
template <typename scalar_t>
__global__ void cond_mul_cuda_forward_wide_kernel(
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds, //indices are in int32 datatype
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> bias,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output) {
    //extern __shared__ uint8_t shared[];
    const int ind = blockIdx.x * blockDim.y + threadIdx.y;
    const int m = weights.size(1);//m input channels
    const int n = weights.size(2);//n output channels
    const int in = threadIdx.x;//index for the output is the thread index

    //Note that storing weights + accumulators in shared memory does not necessarily yield better results.
    //especially if the indices aver very random

    if(ind >= input.size(0)){
        return;
    }

    const int ind_w = inds[ind];
#ifdef FORCE_DOUBLE
	double result = bias[ind_w][0][in];
#else
	scalar_t result = bias[ind_w][0][in];
#endif
    for(int im=0;im<m;im++){
        result += input[ind][im] * weights[ind_w][im][in];
    }
    output[ind][in] = result;
}



/* TODO: optimize once more?
* Memory bandwith is used between 75%(4 consecutive shared weights) and 90% (random weights) at n = 32
* 55 registers for n = 16 56 registers for n = 32
* with shared memory of 4224 bytes for one block there is only a occupancy of 23%
* to improve this one would need to reduce the use of shared memory by a lot
* but even when not adding more register usage the occupancy would not go higher than 50%
* thats due to the 32 threads per block. Warp level synchronization is free though...
* to get full utilization we need to get down to 32 registers per thread & have at least 64 threads per block (two warps)
* most GPUs allow for 64 warps but only 32 blocks to be managed simultaneously
*
* 100% occupancy can only be reached with 64 threads per block or more
* 2048 bytes shared memory ... or less
* and lass than 32 registers utilized
*
* for n==1 we only have 22 registers per thread but the same 23% occupancy and around 50% (not fully random) percent
* of utilized memory bandwith
* TODO:
* Idea to reach better occupancy (50% with 32 threads for one block) for fewer output channels (lets talk about n=1):
* no shared memory usage: reduction after every multiplication (i know this takes 6 cycles with 32, 16, 8, 4, 2, 1
* active threads each). The results would be accumulated up in one result register and wouldn't need any shared memory.
* For n=2 its 2 output registers but the accumulation would only take 5 cycles with 32,16,8,4,2 active threads.
* for n=4 its 4 output registers with only 4 accumulation cycles after each round of multiplication
* For n=8 its 8 output registers (remember its also 8 registers for weights so this might pay off but its already quite
* big
* For n=16 the register count probably would go towards 64 so it could still make sense (50% utilization max) but we have high memory bandwith
* with that already
*
* The issue with n=1 is that we will probably only have 32 inputs and therefore 64 threads per block is hard to achieve
* for n=2 and higher it already is easier to utilize 64 threads even with 32 inputs but still pretty hard to achieve
* the easiest way would be to have multiple of 32 as input and each warp handle one of them.
* The other way to achieve 64 threads would be to have each set of 32 threads work on one pixel... that might be way more efficient
*
* how to use nvidia profiler: (profiling needs sudo rights but python doesn't find modules with sudo)
source venv/bin/activate
sudo env PATH=$PATH nvprof --analysis-metrics -f -o prof.nvvp venv/bin/python test_cuda_cond_mul.py
nvvp prof.nvvp
*/

//TODO: extend this for more than 32 outputs (multiple of 32!) (and template it so we don't loose any  performance    
//Kernel for m multiple of 32 and n being one of 1, 2, 4, 8, 16, 32 TODO: n multiple of 32
//reuse means it is trying not to reload weights at every pixel.
template <typename scalar_t,int m_per_warp,int n_per_set>
__global__ void cond_mul_cuda_forward_deep_reuse32_kernel(
    const int parts_in,
    const int parts_out,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds, //indices are in int32 datatype
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> bias,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output) {
    extern __shared__ uint8_t shared[];

    //TODO: rethink the meaning of threadIdx.x! It should be n

    const int base_ind = 32 * blockIdx.x; // the starting pixel for this block
    const int overall_samples = input.size(0);
    //const int m = weights.size(1); //m... how many input channels alltogether
    //const int n = blockDim.x;//weights.size(2); // should be same asblockDim.x
    //const int in = threadIdx.x;
    const int threads = 32; // threads in one block/warp (always 32)
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int simultaneous_pix = m_per_warp; //threads / n // same as blockDim.y
    const int colums_per_warp = m_per_warp; //threads / n
    const int warps_per_weight_set = n_per_set; // n

    //buffer for the weights of the n outputs (for big n this will use too many registers)
    scalar_t w[n_per_set];

    scalar_t *acc = (scalar_t*)&shared[0];
    //load indices
    int weight_index;
    if( (base_ind + tid) < overall_samples){ //also check if we are overstepping the boundaries
        //load inds for the next 32 pixel
        weight_index = inds[base_ind + tid];
    }
    //if n <=32 (n = n_per_set) we can load enough weights / have enough shared memory for the results of the next 32
    // outputs. If n is multiple of 32 we just do it in multiple blocks/parts.
    for(int l = 0; l < parts_out;l++) {
        //scalar_t v[32];
        //clear the accumulator
        for (int i = 0; i < threads + 1; i++) {
            acc[i * threads + tid] = 0;
        }
        //return;
        //int im = tid;
        //the input/weights of one pixel need to be split into parts
        for (int j = 0; j < parts_in; j++) {
            // load the next 32 values for 32 pixel:
            int last_ind = -1;
            for (int k = 0; k < 32; k++) {
                int pix = base_ind + k;
                if (pix >= overall_samples) {
                    break;
                }
                scalar_t v = input[pix][32 * j + tid];
                int ind_w = __shfl_sync(0xffffffff, weight_index, k);

                // in case of a new index for loading, we reload new weights
                if (ind_w != last_ind) {

                    //we load a set of weights for n_per_set outputs
                    for (int i = 0; i < n_per_set; i++) {
                        int im = j * 32 + i * blockDim.y + threadIdx.y; //index along m direction of weight / input
                        w[i] = weights[ind_w][im][threadIdx.x + l * 32];
                    }
                    last_ind = ind_w;
                }
                scalar_t result = 0;

                //TODO: document this part

                for (int i = 0; i < n_per_set; i++) {
                    result += w[i] *
                              __shfl_sync(0xffffffff, v, i * blockDim.y + threadIdx.y);
                }
                acc[n_per_set * k + threadIdx.y * (32 * n_per_set + n_per_set) + threadIdx.x] += result;

            }
        }
        __syncwarp(); // the warp should be in sync anyway (except for turing gpus... there it might differ!!!)
        // n_per_set also means that one set has the size 32/n_per_set.
        // With 32 threads this means we need n_per_set iterations.
        for (int i = 0; i < n_per_set; i++) {
            int pix_local = i * blockDim.y + threadIdx.y;
            int pix = base_ind + pix_local;
            if (pix >= overall_samples) {
                return;
            }
            int ind_w = __shfl_sync(0xffffffff, weight_index, pix_local);
            scalar_t accu = bias[ind_w][0][threadIdx.x];

            //iterate over all the accumulators for this set of values
            for (int j = 0; j < simultaneous_pix; j++) {
                accu += acc[j * (n_per_set * 32 + n_per_set) +
                            n_per_set * (threadIdx.y + i * blockDim.y) +
                            threadIdx.x]; //the current thread
                //accu +=1;
                /*
                printf("i %d, j %d, thdy %d, thdx %d, accu %f \n",
                            i, j,
                            threadIdx.y, threadIdx.y,
                            acc[ j * (n*32 + n) + n * (threadIdx.y + i * blockDim.y) + threadIdx.x]);
                            */

            }
            output[pix][threadIdx.x + l*32] = accu;

            //TODO: find out if this one is necessary!!
            __syncwarp(); // the warp should be in sync anyway (except for turing gpus... there it might differ!!!)
        }
    }

}

/*
* all what has been written in the comments of the function above is applied here...
* it improves performance for n = 1, 2 and 4 so it's actually pretty useless! For bigger n, it uses too many registers
*/

template <typename scalar_t,int m_per_warp,int n>
__global__ void cond_mul_cuda_forward_deep_reuse32_high_occupancy_kernel(
    const int parts, //the parts in which the input/weights get loaded. for each part all outputs are calculated simultanously
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds, //indices are in int32 datatype
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> bias,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output) {
    extern __shared__ uint8_t shared[];

	//each block has 64 threads in this version, each warp of 32 is processing 32 "pixel"
    const int base_ind = 64 * blockIdx.x + 32*threadIdx.z; // the starting pixel for this block
    const int overall_samples = input.size(0);
    const int m = weights.size(1);
    const int threads = 32; // threads in one block/warp (always 32)
    const int tid = threadIdx.y * blockDim.x + threadIdx.x; //thread index within a warp
    const int simultaneous_pix = m_per_warp; //threads / n // same as blockDim.y
    const int colums_per_warp = m_per_warp; //threads / n
    const int warps_per_weight_set = n; // n
    //const int parts = (m + threads - 1) / threads; //TODO: make this a parameter

    //lets store weights for the processing in a register vairable
    scalar_t w[n];
    //the accumulator should be exactly n*32 long
#ifdef FORCE_DOUBLE
    double *acc = (double*)&shared[sizeof(double) * 32 * n * threadIdx.z];
#else
    scalar_t *acc = (scalar_t*)&shared[sizeof(scalar_t) * 32 * n * threadIdx.z]; //TODO: find out if threadIdx.z really is used!!!
#endif
    //load indices
    int weight_index;
    if( (base_ind + tid) < overall_samples){ //also check if we are overstepping the boundaries
        //load inds for the next 32 pixel
        weight_index = inds[base_ind + tid];
    }
    //scalar_t v[32];
    //clear the accumulator
    for(int i = 0;i < n; i++){
        acc[i * 32 + tid] = 0;
    }


    //the input/weights of each pixel need to be split into parts (of size 32)
    //TODO: think why we are buffering the weights and not input values + output accumulators?
    for(int j = 0; j < parts;j++){
        // load the next 32 values for 32 pixel:
        int last_ind = -1;
        for(int k = 0; k < 32; k++){
            int pix = base_ind + k;

            //stop when we are out of pixel!
            if( pix >= overall_samples){
                break;
            }
            //load the value we need
            scalar_t v = input[pix][32 * j + tid];
            //get the current index we are at from other threads within the warp.(loaded in the beginning)
            int ind_w = __shfl_sync(0xffffffff, weight_index, k);
            if(ind_w != last_ind){
                for(int i=0;i<n;i++){
                	//TODO: is 32 really right?
                	//im consists of a few things
                	//the threadIdx.y... the "lane" of weights/values since only blockDim.y input channels can be processed at once
                	//i... as blockDim.y "lanes" are processed at once a block of 32 needs to be split into n (blockDim.x) parts
                	//j... the part we are at.
                    int im = j * 32 + i * blockDim.y + threadIdx.y; //index along m direction of weight / input
                    w[i] = weights[ind_w][im][threadIdx.x];
                }
                last_ind = ind_w;
            }
            scalar_t result = 0;
            for(int i=0;i<n;i++){
                result += w[i] *
                                __shfl_sync(0xffffffff, v, i * blockDim.y + threadIdx.y);
            }

            //printf("j %d, tid %d, result = %f\n",j,tid, result);
            //now do reduction: I know. for a few clocks this will underutilize the SM
			/*
            if(n <= 16){
                result += __shfl_down_sync(0x0000ffff, result, 16);
                //printf("shfl 16 tid %d, result = %f\n",tid, __shfl_down_sync(0xffffffff, result, 16));
            }
            if(n <= 8 && tid < 16){
                result += __shfl_down_sync(0x000000ff, result, 8);
            }
            if(n <= 4 && tid < 8){
                result += __shfl_down_sync(0x0000000f, result, 4);
            }
            if(n <= 2 && tid < 4){
                result += __shfl_down_sync(0x00000004, result, 2);
            }
            if(n <= 1 && tid < 2){
                result += __shfl_down_sync(0x00000001, result, 1);
            }
            */

			//proper reduction, we don't need to close down threads since they all are synced (the inneccessary additions are not too bad)
			for (int offset = 16; offset >= n; offset /= 2)
				result += __shfl_down_sync(0xffffffff, result, offset);

            if(tid < n){
                //store result in accumulator (shared memory
                acc[tid + k * n] += result;
            }

        }
    }
    __syncwarp(); // the warp should be in sync anyway (except for turing gpus... there it might differ!!!)
	//return;
    for(int i=0;i<n;i++){
        int pix_local = i * blockDim.y + threadIdx.y;
        int pix = base_ind + pix_local;
        if(pix >= overall_samples){
            return;
        }
        int ind_w = __shfl_sync(0xffffffff, weight_index, pix_local);
        output[pix][threadIdx.x] = bias[ind_w][0][threadIdx.x] + acc[pix_local * n + threadIdx.x];
    }

}

__global__ void count_classes(
                const size_t class_count,
                const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds,
                int32_t *counters){
    const int ind = blockIdx.x * blockDim.x + threadIdx.x;

    if(ind >= inds.size(0)){
        return;
    }
    int ind_w = inds[ind];
    if(ind_w >= class_count || ind_w < 0){
        printf("[count_classes]something is seriously off here ind_w %d, class_count %d \n",ind_w, class_count);
    }
    atomicAdd(&counters[ind_w], 1);
}

__global__ void setup_indices(
                const size_t class_count,
                const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds,
                const int32_t *sizes, // the amount of elements on each class
                const int32_t *start_inds, //the staring indices for each class in the lookup buffer
                int32_t *lookup_buffer,
                int32_t *counters
                ){
    const int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if(ind >= inds.size(0)){
        return;
    }
    int ind_w = inds[ind];
    if(ind_w > class_count){
        printf("[setup_indices]something is seriously off here ind_w %d, class_count %d \n",ind_w, class_count);
    }
    int count_old = atomicAdd(&counters[ind_w], 1);
    int start_ind = start_inds[ind_w];
    lookup_buffer[start_ind + count_old] = ind;

}

template <typename scalar_t>
__global__ void cond_mul_cuda_backward_b_kernel(
                        const int32_t *sample_count,
                        const int32_t *starting_inds,
                        const int32_t *lookup_buffer,
                        const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> grad_output,
                        torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> grad_b){
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int im = tid % grad_b.size(2);
    const int ind_w = tid / grad_b.size(2);
    if(ind_w >= grad_b.size(0)){
        return;
    }
#ifdef FORCE_DOUBLE
	double accu = 0;//TODO: get rid of the need for double precision operations here!
#else
	scalar_t accu = 0;
#endif
    const int start_ind = starting_inds[ind_w];
    const int count = sample_count[ind_w];
    for(int i=0; i < count; i++){
        int ind = lookup_buffer[start_ind + i];
        accu += grad_output[ind][im];
    }
    grad_b[ind_w][0][im] = accu;
}


template <typename scalar_t>
__global__ void cond_mul_cuda_backward_w_kernel(
                        const int32_t *sample_count,
                        const int32_t *starting_inds,
                        const int32_t *lookup_buffer,
                        const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> grad_output,
                        const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
                        torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> grad_w){
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int m = grad_w.size(1);
    const int n = grad_w.size(2);
    const int imn = tid % (m * n); // this is not fast or beautiful
    const int ind_w = tid / (m * n);
    const int im = imn / n;
    const int in = imn % n;//this is not beautiful
    if(ind_w >= grad_w.size(0)){
        return;
    }
#ifdef FORCE_DOUBLE
    double accu = 0;//TODO: get rid of the need for double precision operations here!
#else
	scalar_t accu = 0;
#endif
    const int start_ind = starting_inds[ind_w];
    const int count = sample_count[ind_w];
    for(int i=0; i < count; i++){
        int ind = lookup_buffer[start_ind + i];
        // while grad_output will be read relatively efficiently (neighbouring threads read neighbouring values)
        // input is read less efficient. Alltogether it probably is not a superterrible approach, even when
        // the lookup_buffer introduces some pointer chasing.
        // What easily could be improved: Aligning the indices so that workgroups would be working on the same block
        // and have the same count of values to accumulate.
        accu += grad_output[ind][in] * input[ind][im];
    }
    grad_w[ind_w][im][in] = accu;
}

//TODO: why is this commented out? was it because it was too slow?
 /*
template <typename scalar_t>
__global__ void cond_mul_cuda_backward_kernel(
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> d_input,
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> d_weights,
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> d_bias,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> d_output) {

  //batch index
  const int n = blockIdx.y;
  // column index
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c < d_gates.size(2)){
    const auto d_output_gate = tanh(new_cell[n][c]) * grad_h[n][c];
    const auto d_tanh_new_cell = output_gate[n][c] * grad_h[n][c];
    const auto d_new_cell =
        d_tanh(new_cell[n][c]) * d_tanh_new_cell + grad_cell[n][c];


    d_old_cell[n][c] = d_new_cell;
    const auto d_candidate_cell = input_gate[n][c] * d_new_cell;
    const auto d_input_gate = candidate_cell[n][c] * d_new_cell;

    d_gates[n][0][c] =
        d_input_gate * d_sigmoid(gate_weights[n][0][c]);
    d_gates[n][1][c] =
        d_output_gate * d_sigmoid(gate_weights[n][1][c]);
    d_gates[n][2][c] =
        d_candidate_cell * d_elu(gate_weights[n][2][c]);

  }

}
*/
} // namespace

std::vector<torch::Tensor> cond_mul_cuda_forward(
    torch::Tensor input,
    torch::Tensor inds,
    torch::Tensor weights,
    torch::Tensor bias) {
  auto options = torch::TensorOptions()
    .dtype(weights.dtype())
    .layout(torch::kStrided)
    .device(weights.device());
    //.requires_grad(true);
  auto output = torch::zeros({input.size(0), weights.size(2)}, options);

  const int overall_samples = input.size(0);
  const int m = weights.size(1);
  const int n = weights.size(2);
  //std::cout << "i shit you not!!! this is forward" << std::endl;




  AT_DISPATCH_FLOATING_TYPES(input.type(), "cond_mul_forward_cuda", ([&] {
      int threads = m;
      threads = 32;
      if(threads>1024){
        threads = 128;
      }


      int simultaneous_pix = threads/n;
      //memory used for weights, bias, variables and accumulator
      size_t shared_size = 0;

      //TODO: a few issues still reside: for 1 its not better than just having 128 results and then picking the right one
      //also, for 8 its not better than the version without the shared memory
      if((((n == 1) || (n == 2) || (n == 4)) && m%32 == 0) || false){

      	//std::cout << "DEBUG: Running the high occupancy kernel(cond_mul_cuda_forward_deep_reuse32_high_occupancy_kernel)" << std::endl;
            //TODO: reevaluate this implementation!!!!
            //neither is it good for n == 32 nor for n == 16 and for n == 1 its for sure not any better!
#ifdef FORCE_DOUBLE
		  shared_size = 2 * sizeof(double) * 32 * n; // for the accumulator
#else
		  shared_size = 2 * sizeof(scalar_t) * 32 * n; // for the accumulator
#endif

            const int per_group = 32/n;
            const dim3 threads3(n, per_group, 2); //lets have 64 threads per group (doubles the use of shared memory)
            const dim3 blocks((overall_samples + 64 - 1) / 64);
            //std::cout << threads3.x << ", " << threads3.y << ", " << threads3.z << std::endl;

            //std::cout << blocks.x << ", " << blocks.y << ", " << blocks.z << std::endl;
            const int parts = (m + 32 - 1) / 32;

            switch(n){
                case 1:
                    cond_mul_cuda_forward_deep_reuse32_high_occupancy_kernel<scalar_t, 32, 1><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;
                case 2:
                    cond_mul_cuda_forward_deep_reuse32_high_occupancy_kernel<scalar_t, 16, 2><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;
                case 4:
                    cond_mul_cuda_forward_deep_reuse32_high_occupancy_kernel<scalar_t, 8, 4><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;
                case 8:
                    cond_mul_cuda_forward_deep_reuse32_high_occupancy_kernel<scalar_t, 4, 8><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;
                case 16:
                    cond_mul_cuda_forward_deep_reuse32_high_occupancy_kernel<scalar_t, 2, 16><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;
                case 32:
                    cond_mul_cuda_forward_deep_reuse32_high_occupancy_kernel<scalar_t, 1, 32><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;

            }
      //}else if((((n == 8) || (n == 16) || (n == 32) || (n%32 == 0)) && m%32 == 0) && true){
      }else if((((n == 8) || (n == 16) || (n == 32)) && m%32 == 0) && true){
        //std::cout << "DEBUG:Running the deep_reuse kernel(cond_mul_cuda_forward_deep_reuse32_kernel) n,m: " << n << ", " << m << std::endl;
      	//TODO: reevaluate this implementation!!!!


      	//shared memory (approx 4kb used) is used for the accumulator of the weights.
      	//
		shared_size = sizeof(scalar_t) * threads * (threads + 1); // for the accumulator
		//std::cout << "parts: " << parts << ", n: " << n << std::endl;

		const int per_group = 32/n;
		const dim3 threads3(n, per_group);
		const dim3 blocks((overall_samples + 32 - 1) / 32);
		const int parts_in = (m + 32 - 1) / 32;
		const int parts_out = (n - 1)/32 + 1;
		//std::cout << "shared_size: " << shared_size << std::endl;
		switch(n){
		case 1:
			cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, 32, 1><<<blocks, threads3, shared_size>>>(
				parts_in,
				parts_out,
				input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
				inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
				weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
			break;
		case 2:
			cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, 16, 2><<<blocks, threads3, shared_size>>>(
                parts_in,
                parts_out,
				input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
				inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
				weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
			break;
		case 4:
			cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, 8, 4><<<blocks, threads3, shared_size>>>(
                parts_in,
                parts_out,
				input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
				inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
				weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
			break;
		case 8:
			cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, 4, 8><<<blocks, threads3, shared_size>>>(
				parts_in,
				parts_out,
				input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
				inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
				weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
			break;
		case 16:
			cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, 2, 16><<<blocks, threads3, shared_size>>>(
				parts_in,
				parts_out,
				input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
				inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
				weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
			break;
		case 32:
			cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, 1, 32><<<blocks, threads3, shared_size>>>(
				parts_in,
				parts_out,
				input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
				inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
				weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
				output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
			break;
        default:
            //n is multiple of 32, the solution to that is to simply loop this kernel a bit!
            std::cout << "argh! this is a trap" << std::endl;
            /*
            cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, 1, 32><<<blocks, threads3, shared_size>>>(
                    parts_in,
                    parts_out,
                    input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                    inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                    weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                    bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                    output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
		        break;
             */

            }
      }else{
         //version without shared memory
         //TODO: fix the cases in which this fails!!!
         //std::cout << "wide branch" << std::endl;

         size_t per_group = 256/n;// it actually doesn't matter if this were 32 threads. works just the same
         assert(n * per_group == 256);
         const dim3 threads3(n, per_group);
         const dim3 blocks((overall_samples + per_group - 1) / per_group);


         cond_mul_cuda_forward_wide_kernel<scalar_t><<<blocks, threads3>>>(
            input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
            weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
      }
  }));
  //TODO: remove this device synchronize!
  //gpuErrchk( hipPeekAtLastError() );
  //gpuErrchk( hipDeviceSynchronize() );
  return {output};
}


//#define MEASURE_TIME
std::vector<torch::Tensor> cond_mul_cuda_backward(
        torch::Tensor grad_output,//gradient of output
        torch::Tensor input,
        torch::Tensor inds,
        torch::Tensor weights) {
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    //i shit you not its not even trying to compile this!!!!
    auto device = weights.device();
    auto grad_weights = torch::zeros_like(weights);
    auto grad_input = torch::zeros_like(input);
    auto weights_t = torch::transpose(weights, 1, 2);
    auto options =
    torch::TensorOptions()
        .dtype(weights.dtype())
        .layout(torch::kStrided)
        .device(weights.device());
    //.requires_grad(true); //what if the tensor is supposed to be the gradient itself

    auto grad_bias = torch::zeros({weights.size(0), 1, weights.size(2)}, options);//TODO: device of correct type would be nice!!!

    auto bias_back_zero = torch::zeros({weights.size(0), 1, weights.size(1)}, options);
    size_t overall_samples = input.size(0);
    /*
    std::cout << "creating tensors of size = " <<  weights.size(0) * weights.size(2) * sizeof(float) << " and " <<
                weights.size(0) * weights.size(1) * sizeof(float) << " bytes" << std::endl;
    */

    if(grad_output.size(0) == 0){
        //this is not supposed to happen but it happened once so lets keep it here
        assert(0);
    }

    int32_t *sizes_gpu;
    int32_t *starting_inds_gpu;
    int32_t *counters_gpu;
    int32_t *ind_lookup_gpu;
    /*
    std::cout << "allocating temporary " <<
                               sizeof(int32_t) * weights.size(0)*3 +
                               sizeof(int32_t) * grad_output.size(0) << " bytes" << std::endl;*/
    hipMalloc(&sizes_gpu, sizeof(int32_t) * weights.size(0));
    hipMalloc(&starting_inds_gpu, sizeof(int32_t) * weights.size(0));
    hipMalloc(&counters_gpu, sizeof(int32_t) * weights.size(0));
    hipMalloc(&ind_lookup_gpu, sizeof(int32_t) * grad_output.size(0));

    hipMemset(sizes_gpu, 0, sizeof(int32_t) * weights.size(0));
    hipMemset(counters_gpu, 0, sizeof(int32_t) * weights.size(0));
    { //DEBUG: TODO: REMOVE
        //download to cpu
        /*
        std::vector<int32_t> sizes_cpu(weights.size(0));
        hipMemcpy(&sizes_cpu[0], sizes_gpu, sizeof(int32_t) * weights.size(0), hipMemcpyDeviceToHost);

        hipDeviceSynchronize();
        //accumulate the sizes to get the starting positions (on CPU)
        std::vector<int32_t> starting_inds_cpu(weights.size(0));
        int count = 0;
        for(int i=0;i<weights.size(0);i++){
            //std::cout << "sizes_cpu " << sizes_cpu[i] << std::endl;
        }
         */
    }

    //count occurence of each class
    int threads = 256;
    dim3 blocks((overall_samples + threads - 1) / threads);


    count_classes<<<blocks, threads>>>(weights.size(0),//nr of different classes //grad_output.size(0),
                                    inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),
                                    sizes_gpu); //the counts for each class
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		const char* errstr = hipGetErrorString(error);
		std::cout << errstr << std::endl;
	}
	hipDeviceSynchronize();
    //download to cpu
    std::vector<int32_t> sizes_cpu(weights.size(0));
    hipMemcpy(&sizes_cpu[0], sizes_gpu, sizeof(int32_t) * weights.size(0), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    //accumulate the sizes to get the starting positions (on CPU)
    std::vector<int32_t> starting_inds_cpu(weights.size(0));
    int count = 0;


    //std::cout << "calculating the starting positions of " << weights.size(0) << "weights" << std::endl;
    for(int i=0;i<weights.size(0);i++){
        starting_inds_cpu[i] = count;
        //std::cout << "starting_ind " << starting_inds_cpu[i] << std::endl;
        count += sizes_cpu[i];
    }

    if(count != grad_output.size(0)){
		//std::cout << "accumulating weight gradients on gpu: " << gpu << " " << prop.name << std::endl;
        // a serious issue, that needs to be fixed!!!!!
        std::cout << "counted samples " << count << " vs overall samples " << grad_output.size(0) << std::endl;
    }
    assert(count == grad_output.size(0));

    //upload the starting indices for the individual weights
    hipMemcpy(starting_inds_gpu, &starting_inds_cpu[0], sizeof(int32_t) * weights.size(0), hipMemcpyHostToDevice);

    //setup lookup buffer
    setup_indices<<<blocks, threads>>>( weights.size(0),//nr of different classes//grad_output.size(0),
                                    inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),
                                    sizes_gpu,
                                    starting_inds_gpu,
                                    ind_lookup_gpu,
                                    counters_gpu); // the counters for each individiual class

    //Reuse the forward code for the backward pass!!!!
    //TODO: validate this result Otherwise

#ifdef MEASURE_TIME
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Time for preparation: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
    begin = end;
#endif

    std::vector<torch::Tensor>  result =  cond_mul_cuda_forward(
            grad_output,
            inds,
            weights_t,
            bias_back_zero);
    grad_input = result[0];
#ifdef MEASURE_TIME
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    std::cout << "Time for backward: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
    begin = end;
#endif


    //calc gradients for input, w and b
    AT_DISPATCH_FLOATING_TYPES(weights.type(), "cond_mul_forward_cuda", ([&] {
        //gradient for input: (basically the opposite of the forward path with transposed weights and zeroed bias
        const int m = weights_t.size(1);
        const int n = weights_t.size(2);

        //TODO: remove if the validation from up there is fruitful!!! this with the call of the forward pass
        /*
        size_t per_group = 256/std::min(n, 256);//prevent division by zero
        const dim3 threads3(n, per_group);
        dim3 blocks((overall_samples + per_group - 1) / per_group);
        cond_mul_cuda_forward_wide_kernel<scalar_t><<<blocks, threads3>>>(
                    grad_output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(), // input
                    inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                    weights_t.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(), // transposed weights
                    bias_back_zero.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(), // no bias (zero bias)
                    grad_input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>()); //output
        */


        // gradient for b

        threads = 256;
        dim3 blocks((grad_bias.size(0) * grad_bias.size(2) + threads - 1) / threads);
        //blocks.x = (grad_bias.size(0) * grad_bias.size(2) + threads - 1) / threads;
        cond_mul_cuda_backward_b_kernel<scalar_t><<<blocks, threads>>>(
                    sizes_gpu,
                    starting_inds_gpu,
                    ind_lookup_gpu,
                    grad_output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                    grad_bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());


#ifdef MEASURE_TIME
        hipDeviceSynchronize();
        end = std::chrono::steady_clock::now();
        std::cout << "Time for accumulating bias grads: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
        begin = end;
#endif

        threads = 256;
        blocks.x = (grad_weights.size(0) * grad_weights.size(1) * grad_weights.size(2) + threads - 1) / threads;
        cond_mul_cuda_backward_w_kernel<scalar_t><<<blocks, threads>>>(
                    sizes_gpu,
                    starting_inds_gpu,
                    ind_lookup_gpu,
                    grad_output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                    input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                    grad_weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());

    }));

#ifdef MEASURE_TIME
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    std::cout << "Time for accumulating weight grads: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
#endif

    //free all the buffers we created
    hipFree(sizes_gpu);
    hipFree(starting_inds_gpu);
    hipFree(ind_lookup_gpu);
    hipFree(counters_gpu);
    //std::cout << "freeing temporary memory" << std::endl;

  //auto d_gate_weights = d_gates.flatten(1, 2);
  //auto d_weights = d_gate_weights.t().mm(X);
  //auto d_bias = d_gate_weights.sum(/*dim=*/0, /*keepdim=*/true);

  //auto d_X = d_gate_weights.mm(weights);
  //auto d_old_h = d_X.slice(/*dim=*/1, 0, state_size);
  //auto d_input = d_X.slice(/*dim=*/1, state_size);
  /*
	std::cout << "right before the end of backward cond_mul" << std::endl;
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
	std::cout << "after backward cond_mul" << std::endl;
	*/
  return {grad_input, grad_weights, grad_bias};
}
