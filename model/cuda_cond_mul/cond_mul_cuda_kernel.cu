#include "hip/hip_runtime.h"

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>

namespace {
/*
template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(scalar_t z) {
  return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_sigmoid(scalar_t z) {
  const auto s = sigmoid(z);
  return (1.0 - s) * s;
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_tanh(scalar_t z) {
  const auto t = tanh(z);
  return 1 - (t * t);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t elu(scalar_t z, scalar_t alpha = 1.0) {
  return fmaxf(0.0, z) + fminf(0.0, alpha * (exp(z) - 1.0));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_elu(scalar_t z, scalar_t alpha = 1.0) {
  const auto e = exp(z);
  const auto d_relu = z < 0.0 ? 0.0 : 1.0;
  return d_relu + (((alpha * (e - 1.0)) < 0.0) ? (alpha * e) : 0.0);
}
*/
template <typename scalar_t>
__global__ void test_kernel(const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
                            const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds){
    //printf("this is not working\n");
};

//extremely slow kernel... TODO: delete!!!
/*
template <typename scalar_t>
__global__ void cond_mul_cuda_forward_kernel(
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds, //indices are in int32 datatype
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> bias,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output) {
    //printf("i hate you\n");

    const int ind = blockIdx.x * blockDim.x + threadIdx.x;
    const int m = weights.size(1);
    const int n = weights.size(2);
    if(ind >= input.size(0)){
        return;
    }
    int ind_w = inds[ind];
    for(size_t i = 0; i<n ; i++){
        scalar_t accu = bias[ind_w][0][i];

        for(size_t j = 0; j<m; j++){
            //printf("weights %f input %f \n",weights[ind_w][j][i], input[ind][j]);
            accu += weights[ind_w][j][i] * input[ind][j];
        }

        output[ind][i] = accu;

    }
}
*/


template <typename scalar_t>
__global__ void cond_mul_cuda_forward_wide_kernel(
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds, //indices are in int32 datatype
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> bias,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output) {
    //extern __shared__ uint8_t shared[];
    const int ind = blockIdx.x * blockDim.y + threadIdx.y;
    const int m = weights.size(1);
    const int n = weights.size(2);
    const int in = threadIdx.x;

    //scalar_t *v = (scalar_t*)&shared[0 * sizeof(scalar_t)];//TODO: this! if it is necessary
    //int32_t *is = (int32_t*)&shared[n * blockDim.y * sizeof(scalar_t)];

    if(ind >= input.size(0)){
        return;
    }

    const int ind_w = inds[ind];
    scalar_t result = bias[ind_w][0][in];
    for(int im=0;im<m;im++){
        result += input[ind][im] * weights[ind_w][im][in];
    }
    output[ind][in] = result;
}

template <typename scalar_t, bool m_mult_32>
__global__ void cond_mul_cuda_forward_deep_kernel(
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds, //indices are in int32 datatype
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> bias,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output) {
    extern __shared__ uint8_t shared[];

    //TODO: rethink the meaning of threadIdx.x! It should be n

    const int ind = blockIdx.x * blockDim.y;
    const int overall_samples = input.size(0);
    const int m = weights.size(1);
    const int n = blockDim.x;//weights.size(2); // should be same asblockDim.x
    //const int in = threadIdx.x;
    const int threads = blockDim.x*blockDim.y; // threads in one block
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int simultaneous_pix = threads/n; //TODO: template this (should be the same as blockDim.y)
    const int colums_per_warp = threads/n; //TODO: at least replace it with blockDim.y!!!!
    const int warps_per_weight_set = threads / colums_per_warp;
    const int parts = (m + threads -1) / threads; //TODO: make this a parameter


    //int32_t *is = (int32_t*)&shared[0 * sizeof(scalar_t)]; //stored indices (nr of simultaneous points) blockDim.y
    //stored weights for one block of threads n * (threads + 1) // + 1 to fix bank conflict
    scalar_t *w = (scalar_t*)&shared[0];
    //accumulator: (nr of threads + 1 * nr of threads) // + 1 to fix bank conflicts
    scalar_t *acc = (scalar_t*)&shared[(n * (threads + 1) ) * sizeof(scalar_t)];
    //load indices
    int weight_index;
    if( tid < simultaneous_pix &&
        (ind + tid) < overall_samples){ //also check if we are overstepping the boundaries
        //is[tid] = inds[ind + tid];
        weight_index = inds[ind + tid];
    }

    //clear the accumulator
    for(int i = 0;i < threads + 1; i++){
        acc[i * threads + tid] = 0;
    }

    //int im = tid;
    int last_ind = -1; //TODO: incorporate this
    //go trough the pixel that is treated by one warp:
    for(int i = 0;i < simultaneous_pix;i++){
        int pix = ind + i; //pixel index
        //we are hitting the end here:
        if(pix >= overall_samples){
            break;
        }
        //int ind_w = is[i]; //shared memory way of doing things
        int ind_w = __shfl_sync(0xffffffff, weight_index, i);
        //go trough dimension m in the according amount of steps:
        for(int j = 0; j < parts;j++){ //TODO: flip this with the outer loop so we can reuse weights in neighbouring pixel
            //check if we are out of bound
            //load the input vector
            scalar_t v;
            if(threads * j + tid < m || m_mult_32){ // check for bounds (disabled by m_mult_32)
                v = input[pix][threads * j + tid];
            }
            //loading the weights (again in pieces)
            for(int k=0; k < warps_per_weight_set; k++){
                if(j*threads + k * blockDim.y + threadIdx.y < m  || m_mult_32){ //check bounds (disabled by m_mult_32)

                    //n + 1 to fix bank conflict
                     w[k * (threads + 1) + tid] =
                                        weights[ind_w][j * threads + k * blockDim.y + threadIdx.y][threadIdx.x];
                }
            }

            //multiply and store in the according accumulators
            if(threads * j + tid < m  || m_mult_32){//check for bounds (disabled by m_mult_32)
                for(int k=0; k < n;k++){
                    int l = tid * n + k;
                    //bank conflict by reading w fixed by l/threads it makes the access threads + 1 possible
                    acc[(i * n + k) * (threads + 1) + tid] += v * w[l + l / threads];
                }
            }

        }
    }
    {
        //drop out if we are out of bounds:
        int pix = ind + threadIdx.y;
        if(pix >= overall_samples){
            return;
        }

        //int ind_w = is[threadIdx.y]; // will this cause bank conflicts?
        int ind_w = __shfl_sync(0xffffffff, weight_index, threadIdx.y);
        float accu = bias[ind_w][0][threadIdx.x];
        //beautifully sum up the accumulators:
        for(int i = 0; i < threads; i++){
            accu += acc[tid * (threads + 1) + i]; // + 1 for deleting bank conflicts
        }
        //write out the result
        output[pix][threadIdx.x] = accu;

    }

}


template <typename scalar_t, bool m_mult_32>
__global__ void cond_mul_cuda_forward_deep_reuse_kernel(
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds, //indices are in int32 datatype
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> bias,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output) {
    extern __shared__ uint8_t shared[];

    //TODO: rethink the meaning of threadIdx.x! It should be n

    const int ind = blockIdx.x * blockDim.y;
    const int overall_samples = input.size(0);
    const int m = weights.size(1);
    const int n = blockDim.x;//weights.size(2); // should be same asblockDim.x
    //const int in = threadIdx.x;
    const int threads = 32; // threads in one block/warp (always 32)
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int simultaneous_pix = blockDim.y; //threads / n
    const int colums_per_warp = blockDim.y; //threads / n
    const int warps_per_weight_set = n; // n
    const int parts = (m + threads - 1) / threads; //TODO: make this a parameter


    //int32_t *is = (int32_t*)&shared[0 * sizeof(scalar_t)]; //stored indices (nr of simultaneous points) blockDim.y
    //stored weights for one block of threads n * (threads + 1) // + 1 to fix bank conflict
    scalar_t *w = (scalar_t*)&shared[0];
    //accumulator: (nr of threads + 1 * nr of threads) // + 1 to fix bank conflicts
    scalar_t *acc = (scalar_t*)&shared[(n * (threads + 1) ) * sizeof(scalar_t)];
    //load indices
    int weight_index;
    if( tid < simultaneous_pix &&
        (ind + tid) < overall_samples){ //also check if we are overstepping the boundaries
        //is[tid] = inds[ind + tid];
        weight_index = inds[ind + tid];
    }

    //clear the accumulator
    for(int i = 0;i < threads + 1; i++){
        acc[i * threads + tid] = 0;
    }

    //int im = tid;
    //go trough the pixel that is treated by one warp:
    for(int j = 0; j < parts;j++){
        int last_ind = -1;
        for(int i=0;i<simultaneous_pix;i++){
            int pix = ind + i; // pixel index

            if(pix >= overall_samples){
                break;
            }
            int ind_w = __shfl_sync(0xffffffff, weight_index, i);
            scalar_t v;
            if(threads * j + tid < m || m_mult_32){ // check for bounds (disabled by m_mult_32)
                v = input[pix][threads * j + tid];
            }
            if(last_ind != ind_w){ // check if we encountered a new weight
                 //loading the weights (again in pieces)
                for(int k=0; k < warps_per_weight_set; k++){
                    if(j*threads + k * blockDim.y + threadIdx.y < m  || m_mult_32){ //check bounds (disabled by m_mult_32)
                        //n + 1 to fix bank conflict
                         w[k * (threads + 1) + tid] =
                                            weights[ind_w][j * threads + k * blockDim.y + threadIdx.y][threadIdx.x];
                    }
                }
                last_ind = ind_w;
            }

            //multiply and store in the according accumulators
            if(threads * j + tid < m  || m_mult_32){//check for bounds (disabled by m_mult_32)
                for(int k=0; k < n;k++){
                    int l = tid * n + k;
                    //bank conflict by reading w fixed by l/threads it makes the access threads + 1 possible
                    acc[(i * n + k) * (threads + 1) + tid] += v * w[l + l / threads];
                }
            }
        }

    }
    {
        //drop out if we are out of bounds:
        int pix = ind + threadIdx.y;
        if(pix >= overall_samples){
            return;
        }

        //int ind_w = is[threadIdx.y]; // will this cause bank conflicts?
        int ind_w = __shfl_sync(0xffffffff, weight_index, threadIdx.y);
        scalar_t accu = bias[ind_w][0][threadIdx.x];
        //beautifully sum up the accumulators:
        for(int i = 0; i < threads; i++){
            accu += acc[tid * (threads + 1) + i]; // + 1 for deleting bank conflicts
        }
        //write out the result
        output[pix][threadIdx.x] = accu;

    }

}

/* TODO: optimize once more?
* Memory bandwith is used between 75%(4 consecutive shared weights) and 90% (random weights) at n = 32
* 55 registers for n = 16 56 registers for n = 32
* with shared memory of 4224 bytes for one block there is only a occupancy of 23%
* to improve this one would need to reduce the use of shared memory by a lot
* but even when not adding more register usage the occupancy would not go higher than 50%
* thats due to the 32 threads per block. Warp level synchronization is free though...
* to get full utilization we need to get down to 32 registers per thread & have at least 64 threads per block (two warps)
* most GPUs allow for 64 warps but only 32 blocks to be managed simultaneously
*
* 100% occupancy can only be reached with 64 threads per block or more
* 2048 bytes shared memory ... or less
* and lass than 32 registers utilized


* how to use nvidia profiler: (profiling needs sudo rights but python doesn't find modules with sudo)
sudo env PATH=$PATH nvprof --analysis-metrics -f -o prof.nvvp venv/bin/python test_cuda_cond_mul.py
nvvp prof.nvvp
*/
template <typename scalar_t, bool m_mult_32,int m_per_warp,int n>
__global__ void cond_mul_cuda_forward_deep_reuse32_kernel(
    const int parts,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds, //indices are in int32 datatype
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> bias,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output) {
    extern __shared__ uint8_t shared[];

    //TODO: rethink the meaning of threadIdx.x! It should be n

    const int base_ind = 32 * blockIdx.x; // the starting pixel for this block
    const int overall_samples = input.size(0);
    const int m = weights.size(1);
    //const int n = blockDim.x;//weights.size(2); // should be same asblockDim.x
    //const int in = threadIdx.x;
    const int threads = 32; // threads in one block/warp (always 32)
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int simultaneous_pix = m_per_warp; //threads / n // same as blockDim.y
    const int colums_per_warp = m_per_warp; //threads / n
    const int warps_per_weight_set = n; // n
    //const int parts = (m + threads - 1) / threads; //TODO: make this a parameter
    scalar_t w[n];

    scalar_t *acc = (scalar_t*)&shared[0];
    //load indices
    int weight_index;
    if( (base_ind + tid) < overall_samples){ //also check if we are overstepping the boundaries
        //load inds for the next 32 pixel
        weight_index = inds[base_ind + tid];
    }
    //scalar_t v[32];
    //clear the accumulator
    for(int i = 0;i < threads + 1; i++){
        acc[i * threads + tid] = 0;
    }
    //return;
    //int im = tid;
    //the input/weights of one pixel need to be split into parts
    for(int j = 0; j < parts;j++){
        // load the next 32 values for 32 pixel:
        int last_ind = -1;
        for(int k = 0; k < 32; k++){
            int pix = base_ind + k;
            if( pix >= overall_samples){
                break;
            }
            scalar_t v = input[pix][32 * j + tid];
            int ind_w = __shfl_sync(0xffffffff, weight_index, k);
            if(ind_w != last_ind){
                //scalar_t result = 0;//TODO: see if this is a good idea
                for(int i=0;i<n;i++){
                    int im = j * 32 + i * blockDim.y + threadIdx.y; //index along m direction of weight / input
                    w[i] = weights[ind_w][im][threadIdx.x];

                    //TODO: this is not ideal actually!!!! for proper Instruction Level Paralellism we want to have all
                    // the load instructions here and the operations on it later
                    //result += w[i] *//TODO: see if this is a good idea
                    //                __shfl_sync(0xffffffff, v, i * blockDim.y + threadIdx.y);//TODO: see if this is a good idea

                }
                //acc[n * k + threadIdx.y * (32*n + n) + threadIdx.x] += result;//TODO: see if this is a good idea
                last_ind = ind_w;
            }//else{//TODO: see if this is a good idea
                scalar_t result = 0;
                for(int i=0;i<n;i++){
                    //TODO: one could interleave loading w and multiplying
                    result += w[i] *
                                    __shfl_sync(0xffffffff, v, i * blockDim.y + threadIdx.y);
                }
                acc[n * k + threadIdx.y * (32*n + n) + threadIdx.x] += result;
            //}//TODO: see if this is a good idea

        }
        /*
        for(int i = 0;i < threads; i++){
            int pix = base_ind + i;
            if( pix >= overall_samples){
                break;
            }
            v[i] = input[pix][threads * j + tid];
        }
        //each iteration 32/n lines in the mxn matrix are worked on so we need to work n lines to get trough the whole warp:
        for(int i = 0; i < n; i++){
            int last_ind = -1;
            int im = j * 32 + i * blockDim.y + threadIdx.y; //index along m direction of weight / input
            scalar_t w;
            //run trough the 32 consecutive pixel
            for(int k = 0; k < 32;k++){
                if(base_ind + k >= overall_samples){
                    break;
                }
                int ind_w = __shfl_sync(0xffffffff, weight_index, k);
                if(ind_w != last_ind){

                    //printf("im %d, \n",im);
                    w = weights[ind_w][im][threadIdx.x];
                    //w=1;
                    last_ind = ind_w;
                }
                scalar_t result = w *
                            __shfl_sync(0xffffffff, v[k], i * blockDim.y + threadIdx.y);
                //scalar_t vau = __shfl_sync(0xffffffff, v[k], i * blockDim.y + threadIdx.y);
                //fill the accumulator
                //printf("i %d, k %d, thdy %d, thdx %d, result %f, w %f, v %f\n",i, k, threadIdx.y, threadIdx.x, result, w, vau);
                //result = 1;
                acc[n * k + threadIdx.y * (32*n + n) + threadIdx.x] += result;
            }

        }*/
    }
    __syncwarp(); // the warp should be in sync anyway (except for turing gpus... there it might differ!!!)
    for(int i=0;i<n;i++){
        int pix_local = i * blockDim.y + threadIdx.y;
        int pix = base_ind + pix_local;
        if(pix >= overall_samples){
            return;
        }
        int ind_w = __shfl_sync(0xffffffff, weight_index, pix_local);
        scalar_t accu = bias[ind_w][0][threadIdx.x];

        //iterate over all the accumulators for this set of values
        for(int j = 0; j < simultaneous_pix;j++){
            //TODO: why isn't i in this
            accu += acc[ j * (n*32 + n) + n * (threadIdx.y + i * blockDim.y) + threadIdx.x];
            //accu +=1;
            /*
            printf("i %d, j %d, thdy %d, thdx %d, accu %f \n",
                        i, j,
                        threadIdx.y, threadIdx.y,
                        acc[ j * (n*32 + n) + n * (threadIdx.y + i * blockDim.y) + threadIdx.x]);
                        */

        }
        output[pix][threadIdx.x] = accu;
    }

}

template <typename scalar_t, bool m_mult_32>
__global__ void cond_mul_cuda_forward_deep_small_shared_kernel(
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds, //indices are in int32 datatype
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> bias,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output) {
    extern __shared__ uint8_t shared[];

    //TODO: rethink the meaning of threadIdx.x! It should be n

    const int ind = blockIdx.x * blockDim.y;
    const int overall_samples = input.size(0);
    const int m = weights.size(1);
    const int n = blockDim.x;//weights.size(2); // should be same asblockDim.x
    //const int in = threadIdx.x;
    const int threads = blockDim.x*blockDim.y; // threads in one block
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int simultaneous_pix = blockDim.y; //TODO: template this (should be the same as blockDim.y)
    const int colums_per_warp = blockDim.y;
    const int warps_per_weight_set = threads / colums_per_warp;
    const int parts = (m + threads -1) / threads; //TODO: make this a parameter


    scalar_t *acc = (scalar_t*)&shared[0];
    //load indices
    int weight_index;
    if( tid < simultaneous_pix &&
        (ind + tid) < overall_samples){ //also check if we are overstepping the boundaries
        weight_index = inds[ind + tid];
    }
    //clear the accumulator
    for(int i = 0;i < blockDim.y + 1 ; i++){
        acc[i * threads + tid] = 0;
    }
    //int im = tid;
    int last_ind = -1; //TODO: incorporate this
    //go trough the pixel that is treated by one warp:
    for(int i = 0;i < simultaneous_pix;i++){
        int pix = ind + i; //pixel index
        //we are hitting the end here:
        if(pix >= overall_samples){
            break;
        }
        //int ind_w = is[i]; //shared memory way of doing things
        int ind_w = __shfl_sync(0xffffffff, weight_index, i);
        //go trough dimension m in the according amount of steps:
        //scalar_t accu = 0;
        for(int j = 0; j < parts;j++){ //TODO: flip this with the outer loop so we can reuse weights in neighbouring pixel
            //check if we are out of bound
            //load the input vector
            scalar_t v;
            if(threads * j + tid < m || m_mult_32){ // check for bounds (disabled by m_mult_32)
                //v[tid] = input[pix][threads * j + tid]; //TODO: looks like v could be shfl_synced in (if it were not for it being undefined type)
                v = input[pix][threads * j + tid];
            }

            for(int k=0;k< warps_per_weight_set;k++){
                //load weight
                scalar_t w = weights[ind_w][j*threads + k * blockDim.y + threadIdx.y][threadIdx.x];
                //+ blockDim.x to remove bank conflicts
                acc[threadIdx.y*(threads + blockDim.x) + i * blockDim.x + threadIdx.x] +=
                        w *
                         __shfl_sync(0xffffffff, v, k * blockDim.y + threadIdx.y);

            }
            //acc[threadIdx.y*threads + i * blockDim.x + threadIdx.x] += accu;

        }
        //acc[i * threads + tid] = accu; // for each pixel we have a separate accumulator
    }

    //return;
    {
        //drop out if we are out of bounds:
        int pix = ind + threadIdx.y;
        if(pix >= overall_samples){
            return;
        }

        //int ind_w = is[threadIdx.y]; // will this cause bank conflicts?
        int ind_w = __shfl_sync(0xffffffff, weight_index, threadIdx.y);
        float accu = bias[ind_w][0][threadIdx.x];
        //beautifully sum up the accumulators:
        for(int i = 0; i < blockDim.y; i++){
            //+ blockDim.x to remove bank conflicts
            accu += acc[tid + i * (threads + blockDim.x)]; // TODO: no page conflict here but when filling the accumulators
        }
        //write out the result
        output[pix][threadIdx.x] = accu;

    }

}

//this kernel actually is not really faster in many circumstances
/*
template <typename scalar_t>
__global__ void cond_mul_cuda_forward_fast_kernel(
    const size_t group_len,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds, //indices are in int32 datatype
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> bias,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output) {
    extern __shared__ uint8_t shared[];

    int last_ind = -1;
    const int overall_samples = input.size(0);
    const int m = weights.size(1);
    const int n = weights.size(2);
    const int tid = threadIdx.x;
    scalar_t *w = (scalar_t*)&shared[0 * sizeof(scalar_t)]; // size = m * n
    scalar_t *b = (scalar_t*)&shared[(m * n) * sizeof(scalar_t)]; // size = n
    scalar_t *accu = (scalar_t*)&shared[((m * n) + n) * sizeof(scalar_t)]; // size = n * blockDim.x
    scalar_t *values = (scalar_t*)&shared[((m * n) + n + (n * blockDim.x)) * sizeof(scalar_t)]; // size = m

    for(size_t ind = blockIdx.x * group_len; ind < (blockIdx.x + 1) * group_len; ind++){
        if(ind >= overall_samples){
            //end of the dataset is reached
            return;
        }
        int ind_w = inds[ind];
        if(last_ind != ind_w){
            //reloading weights
            int lid = tid; // load index
            while(lid < m*n){
                int im = lid / n; //these two are not efficient either
                int in = lid % n;
                w[in*m + im] = weights[ind_w][im][in]; // this is not super efficient
                lid += blockDim.x;
            }
            //reloading bias:
            lid = tid;
            while(lid < n){
                b[lid] = bias[ind_w][0][lid];
                lid += blockDim.x;
            }
            last_ind = ind_w;
        }

        //loading the input

        int lid = tid;
        while(lid < m){
            values[lid] = input[ind][lid];
            lid += blockDim.x;
        }
        //clear the accumulators:
        lid = tid;
        while(lid < n * blockDim.x){
            accu[lid] = 0.0f;
            lid += blockDim.x;
        }
        __syncthreads();

        //multiply
        //TODO: this is not efficient in case there is more threads than input channels
        lid = tid;
        while(lid < m){
            scalar_t v = values[lid]; //if we read in values here, we actually don't need to use shared memory input[ind][lid];
            for(int i=0 ; i < n; i++){
                accu[i*blockDim.x + tid] += v * w[i*m + lid];
            }
            lid += blockDim.x;
        }

        __syncthreads();
        // reduce
        //TODO: this is not efficient in case there is less output channels than threads (especially if its less than 32)
        lid = tid;
        while(lid < n){
            scalar_t result = b[lid];

            for(int i=0;i<blockDim.x;i++){
                result += accu[lid*blockDim.x + i];
            }
            output[ind][lid] = result;
            lid += blockDim.x;
        }


    }
}
*/

__global__ void count_classes(
                const size_t class_count,
                const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds,
                int32_t *counters){
    const int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if(ind >= inds.size(0)){
        return;
    }
    int ind_w = inds[ind];
    if(ind_w > class_count){
        printf("something is seriously off here ind_w %d, class_count%d \n",ind_w, class_count);
    }
    //printf("result for: %d \n", ind_w);
    atomicAdd(&counters[ind_w], 1);
}

__global__ void setup_indices(
                const size_t class_count,
                const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds,
                const int32_t *sizes, // the amount of elements on each class
                const int32_t *start_inds, //the staring indices for each class in the lookup buffer
                int32_t *lookup_buffer,
                int32_t *counters
                ){
    const int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if(ind >= inds.size(0)){
        return;
    }
    int ind_w = inds[ind];
    if(ind_w > class_count){
        printf("something is seriously off here \n");
    }
    int count_old = atomicAdd(&counters[ind_w], 1);
    int start_ind = start_inds[ind_w];
    lookup_buffer[start_ind + count_old] = ind;

}

template <typename scalar_t>
__global__ void cond_mul_cuda_backward_b_kernel(
                        const int32_t *sample_count,
                        const int32_t *starting_inds,
                        const int32_t *lookup_buffer,
                        const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> grad_output,
                        torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> grad_b){
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int im = tid % grad_b.size(2);
    const int ind_w = tid / grad_b.size(2);
    if(ind_w >= grad_b.size(0)){
        return;
    }
    scalar_t accu = 0;
    const int start_ind = starting_inds[ind_w];
    const int count = sample_count[ind_w];
    for(int i=0; i < count; i++){
        int ind = lookup_buffer[start_ind + i];
        accu += grad_output[ind][im];
    }
    grad_b[ind_w][0][im] = accu;
}


template <typename scalar_t>
__global__ void cond_mul_cuda_backward_w_kernel(
                        const int32_t *sample_count,
                        const int32_t *starting_inds,
                        const int32_t *lookup_buffer,
                        const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> grad_output,
                        const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
                        torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> grad_w){
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int m = grad_w.size(1);
    const int n = grad_w.size(2);
    const int imn = tid % (m * n); // this is not fast or beautiful
    const int ind_w = tid / (m * n);
    const int im = imn / n;
    const int in = imn % n;//this is not beautiful
    if(ind_w >= grad_w.size(0)){
        return;
    }
    scalar_t accu = 0;
    const int start_ind = starting_inds[ind_w];
    const int count = sample_count[ind_w];
    for(int i=0; i < count; i++){
        int ind = lookup_buffer[start_ind + i];
        accu += grad_output[ind][in] * input[ind][im];
    }
    grad_w[ind_w][im][in] = accu;
}
 /*
template <typename scalar_t>
__global__ void cond_mul_cuda_backward_kernel(
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> d_input,
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> d_weights,
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> d_bias,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<int32_t,1,torch::RestrictPtrTraits,size_t> inds,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> weights,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> d_output) {

  //batch index
  const int n = blockIdx.y;
  // column index
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c < d_gates.size(2)){
    const auto d_output_gate = tanh(new_cell[n][c]) * grad_h[n][c];
    const auto d_tanh_new_cell = output_gate[n][c] * grad_h[n][c];
    const auto d_new_cell =
        d_tanh(new_cell[n][c]) * d_tanh_new_cell + grad_cell[n][c];


    d_old_cell[n][c] = d_new_cell;
    const auto d_candidate_cell = input_gate[n][c] * d_new_cell;
    const auto d_input_gate = candidate_cell[n][c] * d_new_cell;

    d_gates[n][0][c] =
        d_input_gate * d_sigmoid(gate_weights[n][0][c]);
    d_gates[n][1][c] =
        d_output_gate * d_sigmoid(gate_weights[n][1][c]);
    d_gates[n][2][c] =
        d_candidate_cell * d_elu(gate_weights[n][2][c]);

  }

}
*/
} // namespace

std::vector<torch::Tensor> cond_mul_cuda_forward(
    torch::Tensor input,
    torch::Tensor inds,
    torch::Tensor weights,
    torch::Tensor bias) {
  auto options = torch::TensorOptions()
    .dtype(weights.dtype())
    .layout(torch::kStrided)
    .device(weights.device());
    //.requires_grad(true);
  auto output = torch::zeros({input.size(0), weights.size(2)}, options);

  const int overall_samples = input.size(0);
  const int m = weights.size(1);
  const int n = weights.size(2);
  //std::cout << "i shit you not!!! this is forward" << std::endl;




  AT_DISPATCH_FLOATING_TYPES(input.type(), "cond_mul_forward_cuda", ([&] {
      int threads = m;
      threads = 32;
      if(threads>1024){
        threads = 128;
      }


      int simultaneous_pix = threads/n;
      //memory used for weights, bias, variables and accumulator
      size_t shared_size =  //sizeof(scalar_t) * (threads) + //store values for each thread TODO: remove
                            //sizeof(int32_t) * simultaneous_pix + //store indices for simultaneous pixel TODO: remove
                            //sizeof(scalar_t) * (n + 1) * threads + //weights for each group of threads TODO: remove
                            sizeof(scalar_t) * n * (threads + 1) + //weights for each group of threads
                            sizeof(scalar_t) * (threads + 1) * threads; // accumulator for the result

      //TODO: a few issues still reside: for 1 its not better than just having 128 results and then picking the right one
      //also, for 8 its not better than the version without the shared memory
      if(((n == 1) || (n == 2) || (n == 4)  || (n == 8) || (n == 16) || (n == 32)) && // maybe templating would work
         m%32 == 0){
            //TODO: reevaluate this implementation!!!!
            //neither is it good for n == 32 nor for n == 16 and for n == 1 its for sure not any better!
            shared_size = sizeof(scalar_t) * threads * (threads + 1); // for the accumulator

            const int per_group = 32/n;
            const dim3 threads3(n, per_group);
            const dim3 blocks((overall_samples + 32 - 1) / 32);
            const int parts = (m + threads - 1) / threads;

            switch(n){
                case 1:
                    cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, true, 32, 1><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;
                case 2:
                    cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, true, 16, 2><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;
                case 4:
                    cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, true, 8, 4><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;
                case 8:
                    cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, true, 4, 8><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;
                case 16:
                    cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, true, 2, 16><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;
                case 32:
                    cond_mul_cuda_forward_deep_reuse32_kernel<scalar_t, true, 1, 32><<<blocks, threads3, shared_size>>>(
                        parts,
                        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                        inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                        weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
                        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
                    break;

            }


      }else if(((n == 1) || (n == 2) || (n == 4)  || (n == 8) || (n == 16)) && // maybe templating would work
                m%32 == 0 &&
                shared_size < 16000){
        //terrible performance at n = 32 (next one is 2 times faster)
        //better performance for n = 16 if the weights are shared for consecutive frames
        //same if they are not shared. Alltogether, it is same / better than
        // cond_mul_cuda_forward_deep_small_shared_kernel which uses shared memory only to a small extend

        //TODO: try kernel with shared weights

        //even though this implementation has quite some bank conflicts it gets by with way less shared memory.
        shared_size = //sizeof(scalar_t) * (threads) + //store values for each thread
                      sizeof(scalar_t) * n * (threads + 1) + //weights for each group of threads
                      sizeof(scalar_t) * (threads + 1) * threads; // accumulator for the result

        const int per_group = 32/n;
        const dim3 threads3(n, per_group);
        const dim3 blocks((overall_samples + per_group - 1) / per_group);
        // the reuse_part only really works well when there is enough
        cond_mul_cuda_forward_deep_reuse_kernel<scalar_t, true><<<blocks, threads3, shared_size>>>(
            input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
            weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
      }else if(((n == 1) || (n == 2) || (n == 4)  || (n == 8) || (n == 16)) && //TODO: find something that works better for 8!!! (templating?)
                m%32 == 0 &&
                shared_size < 16000){
        //version with less use of shared memory
        //same or better than the next one

        //even though this implementation has quite some bank conflicts it gets by with way less shared memory.
        //TODO: check if bank conflicts are still a thing here)
        shared_size = //sizeof(scalar_t) * (threads) + //store values for each thread
                      sizeof(scalar_t) * threads * (threads / n + 1); // accumulator for the result

        const int per_group = 32/n;
        const dim3 threads3(n, per_group);
        const dim3 blocks((overall_samples + per_group - 1) / per_group);

        cond_mul_cuda_forward_deep_small_shared_kernel<scalar_t, true><<<blocks, threads3, shared_size>>>(
            input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
            weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());


      }
      else if(((n == 1) || (n == 2) || (n == 4)  || (n == 8)) &&// || (n == 16)) &&  //TODO: find something that works better for 8!!! (templating?)
         m%32 == 0 &&
         shared_size < 16000){
         //version with shared memory
         //same or batter than the one without shared memory in  only in a few cases

         std::cout << "deep branch with " << shared_size << "bytes of shared memory" << std::endl;

        const int per_group = 32/n;
        const dim3 threads3(n, per_group);
        const dim3 blocks((overall_samples + per_group - 1) / per_group);
        cond_mul_cuda_forward_deep_kernel<scalar_t, true><<<blocks, threads3, shared_size>>>(
            input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
            weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());

      }else{
         //version without shared memory
         //std::cout << "wide branch" << std::endl;

         size_t per_group = 256/n;// it actually doesn't matter if this were 32 threads. works just the same
         const dim3 threads3(n, per_group);

         //shared size could be useful but isn't seemingly
         //shared_size = sizeof(scalar_t) * m * per_group;// + sizeof(int32_t) * per_group;
         const dim3 blocks((overall_samples + per_group - 1) / per_group);
         cond_mul_cuda_forward_wide_kernel<scalar_t><<<blocks, threads3>>>(
            input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
            weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
      }
      /*
      if(n >= 8 || shared_size > 32000){//n%8 == 0){ // maybe this is preferrable for all n bigger than 8 or 16 (not only multiple of n
        std::cout << "wide branch" << std::endl;

        size_t per_group = 256/n;
        const dim3 threads3(n, per_group);

        //shared size could be useful but isn't seemingly
        //shared_size = sizeof(scalar_t) * m * per_group;// + sizeof(int32_t) * per_group;
        const dim3 blocks((overall_samples + per_group - 1) / per_group);
        cond_mul_cuda_forward_wide_kernel<scalar_t><<<blocks, threads3>>>(
            input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
            weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
      }
      else if(shared_size > 48000 && false){ // 48000 is the smalles supported shared memory size (i think)
        //TODO: remove! this is slow
        threads = 1024;
        const dim3 blocks((overall_samples + threads - 1) / threads);
        cond_mul_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
            input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
            weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
      }else if(true){
        std::cout << "deep branch with " << shared_size << "bytes of shared memory" << std::endl;

        const int per_group = 32/n;
        const dim3 threads3(n, per_group);
        const dim3 blocks((overall_samples + per_group - 1) / per_group);
        cond_mul_cuda_forward_deep_kernel<scalar_t><<<blocks, threads3, shared_size>>>(
            input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
            weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());

      }else{
        //TODO: REMOVE! this is slow
        std::cout << "fast branch" << std::endl;
        //the fast branch that tries to utilize shared memory as good as possible
        //unfortunately the fast branch is not really faster in most cases but just in some

        threads = m;
        threads = 32;
        if(threads>1024){
          threads = 128;
        }
        size_t shared_size = sizeof(scalar_t) * (m*n + n + m + n*threads);
        int group_size = 1;
        dim3 blocks(overall_samples);
        if(overall_samples > 128){
            group_size = overall_samples / 128;
            blocks.x = (overall_samples + group_size - 1) / group_size;
        }

        cond_mul_cuda_forward_fast_kernel<scalar_t><<<blocks, threads, shared_size>>>(
            group_size,
            input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
            weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());


      }
      */
  }));

  return {output};
}

std::vector<torch::Tensor> cond_mul_cuda_backward(
        torch::Tensor grad_output,//gradient of output
        torch::Tensor input,
        torch::Tensor inds,
        torch::Tensor weights) {
    //i shit you not its not even trying to compile this!!!!
    auto device = weights.device();
    auto grad_weights = torch::zeros_like(weights);
    auto grad_input = torch::zeros_like(input);
    auto weights_t = torch::transpose(weights, 1, 2);
    auto options =
    torch::TensorOptions()
        .dtype(weights.dtype())
        .layout(torch::kStrided)
        .device(weights.device());
    //.requires_grad(true); //what if the tensor is supposed to be the gradient itself

    auto grad_bias = torch::zeros({weights.size(0), 1, weights.size(2)}, options);//TODO: device of correct type would be nice!!!

    auto bias_back_zero = torch::zeros({weights.size(0), 1, weights.size(1)}, options);
    size_t overall_samples = input.size(0);




    int32_t *sizes_gpu;
    int32_t *starting_inds_gpu;
    int32_t *counters_gpu;
    int32_t *ind_lookup_gpu;
    hipMalloc(&sizes_gpu, sizeof(int32_t) * weights.size(0));
    hipMalloc(&starting_inds_gpu, sizeof(int32_t) * weights.size(0));
    hipMalloc(&counters_gpu, sizeof(int32_t) * weights.size(0));
    hipMalloc(&ind_lookup_gpu, sizeof(int32_t) * grad_output.size(0));

    hipMemset(sizes_gpu, 0, sizeof(int32_t) * weights.size(0));
    hipMemset(counters_gpu, 0, sizeof(int32_t) * weights.size(0));
    { //DEBUG: TODO: REMOVE
        //download to cpu
        std::vector<int32_t> sizes_cpu(weights.size(0));
        hipMemcpy(&sizes_cpu[0], sizes_gpu, sizeof(int32_t) * weights.size(0), hipMemcpyDeviceToHost);

        hipDeviceSynchronize();
        //accumulate the sizes to get the starting positions (on CPU)
        std::vector<int32_t> starting_inds_cpu(weights.size(0));
        int count = 0;
        for(int i=0;i<weights.size(0);i++){
            //std::cout << "sizes_cpu " << sizes_cpu[i] << std::endl;
        }
    }

    //count occurence of each class
    int threads = 256;
    dim3 blocks((overall_samples + threads - 1) / threads);

    count_classes<<<blocks, threads>>>(grad_output.size(0),
                                    inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),
                                    sizes_gpu);

    //download to cpu
    std::vector<int32_t> sizes_cpu(weights.size(0));
    hipMemcpy(&sizes_cpu[0], sizes_gpu, sizeof(int32_t) * weights.size(0), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    //accumulate the sizes to get the starting positions (on CPU)
    std::vector<int32_t> starting_inds_cpu(weights.size(0));
    int count = 0;
    for(int i=0;i<weights.size(0);i++){
        starting_inds_cpu[i] = count;
        //std::cout << "starting_ind " << starting_inds_cpu[i] << std::endl;
        count += sizes_cpu[i];
    }
    if(count != grad_output.size(0)){
        std::cout << "counted samples " << count << " vs overall samples " << grad_output.size(0) << std::endl;
    }
    //TODO: upload accumulated
    hipMemcpy(starting_inds_gpu, &starting_inds_cpu[0], sizeof(int32_t) * weights.size(0), hipMemcpyHostToDevice);

    //setup lookup buffer
    setup_indices<<<blocks, threads>>>( grad_output.size(0),
                                    inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),
                                    sizes_gpu,
                                    starting_inds_gpu,
                                    ind_lookup_gpu,
                                    counters_gpu);


    //TODO: calc gradients for input, w and b
    AT_DISPATCH_FLOATING_TYPES(weights.type(), "cond_mul_forward_cuda", ([&] {
        //gradient for input: (basically the opposite of the forward path with transposed weights and zeroed bias
        const int m = weights_t.size(1);
        const int n = weights_t.size(2);
        size_t per_group = 256/n;
        const dim3 threads3(n, per_group);

        //shared size could be useful but isn't seemingly
        //shared_size = sizeof(scalar_t) * m * per_group;// + sizeof(int32_t) * per_group;
        dim3 blocks((overall_samples + per_group - 1) / per_group);
        cond_mul_cuda_forward_wide_kernel<scalar_t><<<blocks, threads3>>>(
                    grad_output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(), // input
                    inds.packed_accessor<int32_t,1,torch::RestrictPtrTraits,size_t>(),//indices are in cheaper datatype
                    weights_t.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(), // transposed weights
                    bias_back_zero.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(), // no bias (zero bias)
                    grad_input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>()); //output


        // gradient for b
        /*
        cond_mul_cuda_backward_b_kernel<<<blocks, threads3>>>(
                        const int32_t *sample_count,
                        const int32_t *starting_inds,
                        const int32_t *lookup_buffer,
                        const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> grad_output,
                        torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> grad_b){
                        */
                        //TODO: reinsert

        threads = 256;
        blocks.x = (grad_bias.size(0) * grad_bias.size(2) + threads - 1) / threads;
        cond_mul_cuda_backward_b_kernel<scalar_t><<<blocks, threads>>>(
                    sizes_gpu,
                    starting_inds_gpu,
                    ind_lookup_gpu,
                    grad_output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                    grad_bias.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());



        threads = 256;
        blocks.x = (grad_weights.size(0) * grad_weights.size(1) * grad_weights.size(2) + threads - 1) / threads;
        cond_mul_cuda_backward_w_kernel<scalar_t><<<blocks, threads>>>(
                    sizes_gpu,
                    starting_inds_gpu,
                    ind_lookup_gpu,
                    grad_output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                    input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
                    grad_weights.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());


        //gradient for w
        /*
        blocks.x = grad_bias.size
        __global__ void cond_mul_cuda_backward_w_kernel(
                        const int32_t *class_count,
                        const int32_t *sizes,
                        const int32_t *lookup_buffer,
                        const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> grad_output,
                        const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
                        torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> grad_w){ */

    }));




    //free all the buffers we created
    hipFree(sizes_gpu);
    hipFree(starting_inds_gpu);
    hipFree(ind_lookup_gpu);
    hipFree(counters_gpu);

  //auto d_gate_weights = d_gates.flatten(1, 2);
  //auto d_weights = d_gate_weights.t().mm(X);
  //auto d_bias = d_gate_weights.sum(/*dim=*/0, /*keepdim=*/true);

  //auto d_X = d_gate_weights.mm(weights);
  //auto d_old_h = d_X.slice(/*dim=*/1, 0, state_size);
  //auto d_input = d_X.slice(/*dim=*/1, state_size);

  return {grad_input, grad_weights, grad_bias};
}
